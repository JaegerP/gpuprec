#include "hip/hip_runtime.h"
#ifndef BENCHMARK_CU
#define BENCHMARK_CU


#include <cstdlib>
#include <cstdio>
#include <cstdarg>
#include <iostream>
#include <qd/qd_real.h>
#include <qd/fpu.h>
#include <omp.h>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <vector>
#include <memory>


#include "gqd.cu"
#include "cuda_util.h"
#include "cuda_util.h"
#include "test_util.h"
#include "test_common.h"
#include "gqdtest.h"

using namespace std;


/* general macro utilities */
#define FUNC_START_MSG printf("%s start ............................................\n", __func__);
#define FUNC_END_MSG   printf("%s done  ...........................................\n\n", __func__);

template<class c_t, class g_t>
void test_sqrt(const unsigned int numElement) {

    FUNC_START_MSG;

    c_t* dd_in = new c_t[numElement];
    c_t* gold_out = new c_t[numElement];
    c_t low = "0.0";
    c_t high = "1.0";
    randArray(dd_in, numElement, low, high);
    g_t* gdd_in = new g_t[numElement];
    g_t* gdd_out = new g_t[numElement];
    qd2gqd(dd_in, gdd_in, numElement);


    unsigned int numBlock = 128;
    unsigned int numThread = 128;
    device_math(gdd_in, numElement, gdd_out, SQRT, numBlock, numThread);
    c_t* gpu_out = new c_t[numElement];
    gqd2qd(gdd_out, gpu_out, numElement);

    INIT_TIMER;
    START_TIMER;
#pragma omp parallel for
    for (unsigned int i = 0; i < numElement; i++) {
        gold_out[i] = sqrt(dd_in[i]);
    }
    END_TIMER;
    PRINT_TIMER_SEC("CPU sqrt");

    checkTwoArray(gold_out, gpu_out, numElement);

    delete[] dd_in;
    delete[] gold_out;
    delete[] gdd_in;
    delete[] gdd_out;
    delete[] gpu_out;

    FUNC_END_MSG;
}

template<class c_t, class g_t>
void test_exp(const unsigned int numElement) {

    FUNC_START_MSG;


    c_t* dd_in = new c_t[numElement];
    c_t* gold_out = new c_t[numElement];
    c_t low = "0.0";
    c_t high = "1.0";
    randArray(dd_in, numElement, low, high);
    g_t* gdd_in = new g_t[numElement];
    g_t* gdd_out = new g_t[numElement];
    qd2gqd(dd_in, gdd_in, numElement);


    unsigned int numBlock = 128;
    unsigned int numThread = 128;
    device_math(gdd_in, numElement, gdd_out, EXP, numBlock, numThread);
    c_t* gpu_out = new c_t[numElement];
    gqd2qd(gdd_out, gpu_out, numElement);

    INIT_TIMER;
    START_TIMER;
#pragma omp parallel for
    for (unsigned int i = 0; i < numElement; i++) {
        gold_out[i] = exp(dd_in[i]);
    }
    END_TIMER;
    PRINT_TIMER_SEC("CPU exp");

    checkTwoArray(gold_out, gpu_out, numElement);

    delete[] dd_in;
    delete[] gold_out;
    delete[] gdd_in;
    delete[] gdd_out;
    delete[] gpu_out;

    FUNC_END_MSG;
}

template<class c_t, class g_t>
void test_log(const unsigned int numElement) {

    FUNC_START_MSG;

    c_t* dd_in = new c_t[numElement];
    c_t* gold_out = new c_t[numElement];
    c_t low = "0.0";
    c_t high = "1.0";
    randArray(dd_in, numElement, low, high);
    g_t* gdd_in = new g_t[numElement];
    g_t* gdd_out = new g_t[numElement];
    qd2gqd(dd_in, gdd_in, numElement);


    unsigned int numBlock = 128;
    unsigned int numThread = 128;
    device_math(gdd_in, numElement, gdd_out, LOG, numBlock, numThread);
    c_t* gpu_out = new c_t[numElement];
    gqd2qd(gdd_out, gpu_out, numElement);


    INIT_TIMER;
    START_TIMER;
#pragma omp parallel for
    for (unsigned int i = 0; i < numElement; i++) {
        gold_out[i] = log(dd_in[i]);
    }
    END_TIMER;
    PRINT_TIMER_SEC("CPU log");

    checkTwoArray(gold_out, gpu_out, numElement);

    delete[] dd_in;
    delete[] gold_out;
    delete[] gdd_in;
    delete[] gdd_out;
    delete[] gpu_out;

    FUNC_END_MSG;
}

template<class c_t, class g_t>
void test_sin(const unsigned int numElement) {

    FUNC_START_MSG;

    c_t* dd_in = new c_t[numElement];
    c_t* gold_out = new c_t[numElement];
    c_t low = "0.0";
    c_t high = "1.0";
    randArray(dd_in, numElement, low, high);
    g_t* gdd_in = new g_t[numElement];
    g_t* gdd_out = new g_t[numElement];
    qd2gqd(dd_in, gdd_in, numElement);


    unsigned int numBlock = 128;
    unsigned int numThread = 128;
    device_math(gdd_in, numElement, gdd_out, SIN, numBlock, numThread);
    c_t* gpu_out = new c_t[numElement];
    gqd2qd(gdd_out, gpu_out, numElement);


    INIT_TIMER;
    START_TIMER;
#pragma omp parallel for
    for (unsigned int i = 0; i < numElement; i++) {
        gold_out[i] = sin(dd_in[i]);
    }
    END_TIMER;
    PRINT_TIMER_SEC("CPU sin");

    checkTwoArray(gold_out, gpu_out, numElement);

    delete[] dd_in;
    delete[] gold_out;
    delete[] gdd_in;
    delete[] gdd_out;
    delete[] gpu_out;

    FUNC_END_MSG;
}

template<class c_t, class g_t>
void test_acos(const unsigned int numElement) {

    FUNC_START_MSG;

    c_t* dd_in = new c_t[numElement];
    c_t* gold_out = new c_t[numElement];
    c_t low = "-1.0";
    c_t high = "1.0";
    randArray(dd_in, numElement, low, high);
    g_t* gdd_in = new g_t[numElement];
    g_t* gdd_out = new g_t[numElement];
    qd2gqd(dd_in, gdd_in, numElement);


    unsigned int numBlock = 128;
    unsigned int numThread = 128;
    device_math(gdd_in, numElement, gdd_out, ACOS, numBlock, numThread);
    c_t* gpu_out = new c_t[numElement];
    gqd2qd(gdd_out, gpu_out, numElement);


    INIT_TIMER;
    START_TIMER;
#pragma omp parallel for
    for (unsigned int i = 0; i < numElement; i++) {
        gold_out[i] = acos(dd_in[i]);
    }
    END_TIMER;
    PRINT_TIMER_SEC("CPU acos");

    checkTwoArray(gold_out, gpu_out, numElement);

    delete[] dd_in;
    delete[] gold_out;
    delete[] gdd_in;
    delete[] gdd_out;
    delete[] gpu_out;

    FUNC_END_MSG;
}

template<class c_t, class g_t>
void test_asin(const unsigned int numElement) {

    FUNC_START_MSG;

    c_t* dd_in = new c_t[numElement];
    c_t* gold_out = new c_t[numElement];
    c_t low = "-1.0";
    c_t high = "1.0";
    randArray(dd_in, numElement, low, high);
    g_t* gdd_in = new g_t[numElement];
    g_t* gdd_out = new g_t[numElement];
    qd2gqd(dd_in, gdd_in, numElement);


    unsigned int numBlock = 128;
    unsigned int numThread = 128;
    device_math(gdd_in, numElement, gdd_out, ASIN, numBlock, numThread);
    c_t* gpu_out = new c_t[numElement];
    gqd2qd(gdd_out, gpu_out, numElement);


    INIT_TIMER;
    START_TIMER;
#pragma omp parallel for
    for (unsigned int i = 0; i < numElement; i++) {
        gold_out[i] = asin(dd_in[i]);
    }
    END_TIMER;
    PRINT_TIMER_SEC("CPU asin");

    checkTwoArray(gold_out, gpu_out, numElement);

    delete[] dd_in;
    delete[] gold_out;
    delete[] gdd_in;
    delete[] gdd_out;
    delete[] gpu_out;

    FUNC_END_MSG;
}

template<class c_t, class g_t>
void test_cos(unsigned int numElement) {

    FUNC_START_MSG;

    c_t* dd_in = new c_t[numElement];
    c_t* gold_out = new c_t[numElement];
    c_t low = "0.0";
    c_t high = "1.0";
    randArray(dd_in, numElement, low, high);
    g_t* gdd_in = new g_t[numElement];
    g_t* gdd_out = new g_t[numElement];
    qd2gqd(dd_in, gdd_in, numElement);


    unsigned int numBlock = 128;
    unsigned int numThread = 128;
    device_math(gdd_in, numElement, gdd_out, COS, numBlock, numThread);
    c_t* gpu_out = new c_t[numElement];
    gqd2qd(gdd_out, gpu_out, numElement);


    INIT_TIMER;
    START_TIMER;
#pragma omp parallel for
    for (unsigned int i = 0; i < numElement; i++) {
        gold_out[i] = cos(dd_in[i]);
    }
    END_TIMER;
    PRINT_TIMER_SEC("CPU cos");

    checkTwoArray(gold_out, gpu_out, numElement);

    delete[] dd_in;
    delete[] gold_out;
    delete[] gdd_in;
    delete[] gdd_out;
    delete[] gpu_out;

    FUNC_END_MSG;
}

template<class c_t, class g_t>
void test_tan(unsigned int numElement) {

    FUNC_START_MSG;

    c_t* dd_in = new c_t[numElement];
    c_t* gold_out = new c_t[numElement];
    c_t low = "0.0";
    c_t high = "1.0";
    randArray(dd_in, numElement, low, high);
    g_t* gdd_in = new g_t[numElement];
    g_t* gdd_out = new g_t[numElement];
    qd2gqd(dd_in, gdd_in, numElement);


    unsigned int numBlock = 128;
    unsigned int numThread = 128;
    device_math(gdd_in, numElement, gdd_out, TAN, numBlock, numThread);
    c_t* gpu_out = new c_t[numElement];
    gqd2qd(gdd_out, gpu_out, numElement);


    INIT_TIMER;
    START_TIMER;
#pragma omp parallel for
    for (unsigned int i = 0; i < numElement; i++) {
        gold_out[i] = tan(dd_in[i]);
    }
    END_TIMER;
    PRINT_TIMER_SEC("CPU tan");

    checkTwoArray(gold_out, gpu_out, numElement);

    delete[] dd_in;
    delete[] gold_out;
    delete[] gdd_in;
    delete[] gdd_out;
    delete[] gpu_out;

    FUNC_END_MSG;
}

/*
template<class c_t, class g_t>
void test_atan(unsigned int numElement) {

        FUNC_START_MSG;

        c_t* dd_in = new c_t[numElement];
        c_t* gold_out = new c_t[numElement];
        c_t low = "0.0";
        c_t high = "1.0";
        randArray(dd_in, numElement, low, high);
        g_t* gdd_in = new g_t[numElement];
        g_t* gdd_out = new g_t[numElement];
        qd2gqd(dd_in, gdd_in, numElement);


        unsigned int numBlock = 128;
        unsigned int numThread = 128;
        device_math(gdd_in, numElement, gdd_out, ATAN, numBlock, numThread);
        c_t* gpu_out = new c_t[numElement];
        gqd2qd(gdd_out, gpu_out, numElement);


        INIT_TIMER;
        START_TIMER;
#pragma omp parallel for
        for(unsigned int i = 0; i < numElement; i++) {
                gold_out[i] = atan(dd_in[i]);
        }
        END_TIMER;
        PRINT_TIMER_SEC("CPU tan");

        checkTwoArray(gold_out, gpu_out, numElement);

        delete[] dd_in;
        delete[] gold_out;
        delete[] gdd_in;
        delete[] gdd_out;
        delete[] gpu_out;

        FUNC_END_MSG;
}
 */


template<class c_t, class g_t>
void test_add(const unsigned int numElement) {

    FUNC_START_MSG;

    c_t* dd_in1 = new c_t[numElement];
    c_t* dd_in2 = new c_t[numElement];
    c_t* gold_out = new c_t[numElement];
    c_t low = "-1.0";
    c_t high = "1.0";
    randArray(dd_in1, numElement, low, high, 777);
    randArray(dd_in2, numElement, low, high, 888);
    g_t* gdd_in1 = new g_t[numElement];
    g_t* gdd_in2 = new g_t[numElement];
    g_t* gdd_out = new g_t[numElement];
    qd2gqd(dd_in1, gdd_in1, numElement);
    qd2gqd(dd_in2, gdd_in2, numElement);


    unsigned int numBlock = 128;
    unsigned int numThread = 128;
    device_basic(gdd_in1, gdd_in2, gdd_out, numElement, ADD, numBlock, numThread);
    c_t* gpu_out = new c_t[numElement];
    gqd2qd(gdd_out, gpu_out, numElement);

    INIT_TIMER;
    START_TIMER;
#pragma omp parallel for
    for (unsigned int i = 0; i < numElement; i++) {
        gold_out[i] = dd_in1[i] + dd_in2[i];
    }
    END_TIMER;
    PRINT_TIMER_SEC("CPU add");

    checkTwoArray(gold_out, gpu_out, numElement);

    delete[] dd_in1;
    delete[] dd_in2;
    delete[] gold_out;
    delete[] gdd_in1;
    delete[] gdd_in2;
    delete[] gdd_out;
    delete[] gpu_out;

    FUNC_END_MSG;
}

template<class c_t, class g_t>
void test_mul(const unsigned int numElement) {

    FUNC_START_MSG;

    c_t* dd_in1 = new c_t[numElement];
    c_t* dd_in2 = new c_t[numElement];
    c_t* gold_out = new c_t[numElement];
    c_t low = "-1.0";
    c_t high = "1.0";
    randArray(dd_in1, numElement, low, high, 777);
    randArray(dd_in2, numElement, low, high, 888);

    g_t* gdd_in1 = new g_t[numElement];
    g_t* gdd_in2 = new g_t[numElement];
    g_t* gdd_out = new g_t[numElement];
    qd2gqd(dd_in1, gdd_in1, numElement);
    qd2gqd(dd_in2, gdd_in2, numElement);


    unsigned int numBlock = 128;
    unsigned int numThread = 128;
    device_basic(gdd_in1, gdd_in2, gdd_out, numElement, MUL, numBlock, numThread);
    c_t* gpu_out = new c_t[numElement];
    gqd2qd(gdd_out, gpu_out, numElement);

    INIT_TIMER;
    START_TIMER;
#pragma omp parallel for
    for (unsigned int i = 0; i < numElement; i++) {
        gold_out[i] = dd_in1[i] * dd_in2[i];
    }
    END_TIMER;
    PRINT_TIMER_SEC("CPU mul");

    checkTwoArray(gold_out, gpu_out, numElement);

    delete[] dd_in1;
    delete[] dd_in2;
    delete[] gold_out;
    delete[] gdd_in1;
    delete[] gdd_in2;
    delete[] gdd_out;
    delete[] gpu_out;

    FUNC_END_MSG;
}

template<class c_t, class g_t>
void test_div(const unsigned int numElement) {

    FUNC_START_MSG;

    c_t* dd_in1 = new c_t[numElement];
    c_t* dd_in2 = new c_t[numElement];
    c_t* gold_out = new c_t[numElement];
    c_t low = "-1.0";
    c_t high = "1.0";
    randArray(dd_in1, numElement, low, high, 777);
    randArray(dd_in2, numElement, low, high, 888);
    g_t* gdd_in1 = new g_t[numElement];
    g_t* gdd_in2 = new g_t[numElement];
    g_t* gdd_out = new g_t[numElement];
    qd2gqd(dd_in1, gdd_in1, numElement);
    qd2gqd(dd_in2, gdd_in2, numElement);


    unsigned int numBlock = 128;
    unsigned int numThread = 128;
    device_basic(gdd_in1, gdd_in2, gdd_out, numElement, DIV, numBlock, numThread);
    c_t* gpu_out = new c_t[numElement];
    gqd2qd(gdd_out, gpu_out, numElement);

    INIT_TIMER;
    START_TIMER;
#pragma omp parallel for
    for (unsigned int i = 0; i < numElement; i++) {
        gold_out[i] = dd_in1[i] / dd_in2[i];
    }
    END_TIMER;
    PRINT_TIMER_SEC("CPU div");

    checkTwoArray(gold_out, gpu_out, numElement);

    delete[] dd_in1;
    delete[] dd_in2;
    delete[] gold_out;
    delete[] gdd_in1;
    delete[] gdd_in2;
    delete[] gdd_out;
    delete[] gpu_out;

    FUNC_END_MSG;
}

/*
int main(int argc, char** argv) {
    const int omp_num_thread = 16;
    omp_set_num_threads(omp_num_thread);
    printf("omp_num_thread = %d\n", omp_num_thread);

    unsigned int old_cw;
    fpu_fix_start(&old_cw);



    printf("==================================================================\n");
    printf("******************** double-double precision *********************\n");
    printf("==================================================================\n");
    GDDStart();
    unsigned int numElement = 10000000;
    printf("numElement = %d\n", numElement);
    test_add<dd_real, gdd_real>(numElement);
    test_mul<dd_real, gdd_real>(numElement);
    test_div<dd_real, gdd_real>(numElement);
    test_sqrt<dd_real, gdd_real>(numElement);
    test_exp<dd_real, gdd_real>(numElement);
    test_log<dd_real, gdd_real>(numElement);
    test_sin<dd_real, gdd_real>(numElement);
    test_acos<dd_real, gdd_real>(numElement);
    test_asin<dd_real, gdd_real>(numElement);
    test_tan<dd_real, gdd_real>(numElement);
    GDDEnd();

    printf("\n\n");

    printf("==================================================================\n");
    printf("********************* quad-double precision **********************\n");
    printf("==================================================================\n");
    GQDStart();
    numElement = 1000000;
    printf("numElement = %d\n", numElement);
    test_add<qd_real, gqd_real>(numElement);
    test_mul<qd_real, gqd_real>(numElement);
    test_div<qd_real, gqd_real>(numElement);
    test_sqrt<qd_real, gqd_real>(numElement);
    test_exp<qd_real, gqd_real>(numElement);
    test_log<qd_real, gqd_real>(numElement);
    test_sin<qd_real, gqd_real>(numElement);
    test_tan<qd_real, gqd_real>(numElement);
    GQDEnd();

    fpu_fix_end(&old_cw);
    return EXIT_SUCCESS;
}

*/

enum OP_TYPE {UNARY = 1, BINARY = 2};

template<class GPU_T>
void testFunc(const int numElement, OP_TYPE type, ...) {
	cout << "numElement: " << numElement << endl;
    cout << "op type: " << type << endl;
    va_list vl;

	va_start(vl, type);

	if (UNARY == type) {
		GPU_T a = va_arg(vl, GPU_T);
		cout << "a = " << a << endl;
	} else if(BINARY == type) {
		GPU_T a = va_arg(vl, GPU_T);
		GPU_T b = va_arg(vl, GPU_T);
		cout << "a = " << a << endl;
		cout << "b = " << b << endl;
	} else {
		throw std::runtime_error("UNKNOW OP_TYPE");
	}
 
    va_end(vl);
}


template<typename T_R, typename T_A, typename T_B>
__device__
T_R op_add(const T_A& a, const T_B& b) {
	return a + b;
}

template<typename T_R, typename T_A>
__host__ __device__
T_R op_exp(const T_A a) {
	return exp(a);
}


typedef gdd_real (*gdd_unary_func)(gdd_real);
typedef dd_real (*dd_unary_func)(dd_real);

__device__ gdd_unary_func gdd_pexp = op_exp<gdd_real, gdd_real>;

template<typename T_OUT, typename T_IN1, class T_OP>
__global__
void benchmark_kernel(T_OUT* d_out, T_IN1* d_in1, T_OP* op, const int numElement) {
	const int numTotalThread = blockDim.x*gridDim.x;
	const int threadOffset = blockIdx.x*blockDim.x + threadIdx.x;

	for(int i = threadOffset; i < numElement; i += numTotalThread) {
		d_out[i] = op[0](d_in1[i]);
	}
}


template<class OP>
void host_kernel(gdd_real* out, gdd_real* in1, OP op, const int numElement) {
	dd_real* t_out = new dd_real[numElement];
	dd_real* t_in1 = new dd_real[numElement];

	gqd2qd(in1, t_in1, numElement);
	for(int i = 0; i < numElement; i += 1) {
		t_out[i] = op(t_in1[i]);
	}
	qd2gqd(t_out, out, numElement);

	delete[] t_out;
	delete[] t_in1;
}



template<typename T_OUT, typename T_IN1, class T_H_OP, class T_D_OP>
void benchmark(const int numElement, T_H_OP h_op, T_D_OP &d_op) {
	// Allocate host memory for operands
	T_IN1* h_in1 = new T_IN1[numElement];

	// Generate randome numbers [-1, 1] for the operands on the host
	randArray(h_in1, numElement, -1.0, 1.0);	

	// Copy the input data to the device
	T_IN1* d_in1 = NULL;
	GPUMALLOC((void**)&d_in1, sizeof(T_IN1)*numElement);
	TOGPU(d_in1, h_in1, sizeof(T_IN1)*numElement);

	// Allocate memory for results on the device 
	T_OUT* d_out = NULL;
	GPUMALLOC((void**)&d_out, sizeof(T_OUT)*numElement);

	// Assign the device function pointer
	T_D_OP* h_f = (T_D_OP*)malloc(sizeof(T_D_OP));
	T_D_OP* d_f = NULL;
	GPUMALLOC((void**)&d_f, sizeof(T_D_OP));
	checkCudaErrors(hipMemcpyFromSymbol(h_f, HIP_SYMBOL(d_op), sizeof(T_D_OP)));
	checkCudaErrors(hipMemcpy(d_f, h_f, sizeof(T_D_OP), hipMemcpyHostToDevice));
	cout << "Function pointer assignment done" << endl;

	// Performance computation on device
	benchmark_kernel<T_OUT, T_IN1, T_D_OP><<<512, 512>>>(d_out, d_in1, d_f, numElement);
	getLastCudaError("benchmark_kernel");
	cout << "Computation on device done" << endl;

	// Copy result back from the GPU
	T_OUT* h_out = new T_OUT[numElement];	
	FROMGPU(h_out, d_out, sizeof(T_OUT)*numElement);

	// Performance computation on host
	T_OUT* gold_out = new T_OUT[numElement];
	host_kernel(gold_out, h_in1, h_op, numElement);
	cout << "Computation on host done" << endl;
	
	// Check results
	for(int i = 0; i < numElement; i += 1) {
		cout << h_out[i] << ", " << gold_out[i] << endl;
	}

	// Memory cleanup
	delete[] h_in1;
	delete[] h_out;
	GPUFREE(d_in1);
	GPUFREE(d_out);
}


int main(int argc, char** argv) {
    unsigned int old_cw;

	// Turn on
    fpu_fix_start(&old_cw);
	GDDStart();	


    printf("==================================================================\n");
    printf("******************** double-double precision *********************\n");
    printf("==================================================================\n");
    
	const int numElement = 10;
	benchmark<gdd_real, gdd_real>(numElement, &op_exp<dd_real, dd_real>, gdd_pexp);

	// Shutdown
    GQDEnd();
    fpu_fix_end(&old_cw);

	return EXIT_SUCCESS;
}


#endif /*BENCHMARK_CU*/
